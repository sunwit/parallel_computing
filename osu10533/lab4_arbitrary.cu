/*
   Name: Wei Sun (sun.1868@osu.edu)
   Class Section: CSE 5441 AU2018(27372) W F 12:45pm - 2:00pm
   Assignment Number: Programming Assignment 4
   Summary: Producer Consumer problem with GPU programming
   Submission: AU18
   this is the arbitrary version of producer-consumer with GPU programming. 
   the most time consuming part of producer and consumer is transforms. 
   then, we let GPU to do this most time consuming part. but, as you can see 
   we do these transforms in an arbitary order. so, version 1 is less efficient
   than version 2 which can reorder the transform part for GPU. 
*/


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<string.h>
#include<inttypes.h>
#include<time.h>

#define BUF_SIZE 5  //queue size is 5

struct work_entry //struct of queue
{
   char cmd;  // save letters
   uint16_t pkey; //save the key before producer transforms it
   uint16_t akey; //save the key after consumer transforms it

};


__device__ uint16_t transformA(uint16_t x); //transformA 
__device__ uint16_t transformB(uint16_t x); //transformB
__device__ uint16_t transformC(uint16_t x); //transformC
__device__ uint16_t transformD(uint16_t x); //transformD


__global__ void transformerPro(struct work_entry *d_a)  // producer transforms in GPU
{
      int idx = threadIdx.x;
      char cm = d_a[idx].cmd;
      uint16_t ke;
      if(cm=='A') ke=transformA(d_a[idx].pkey);
      else if(cm=='B') ke=transformB(d_a[idx].pkey);
      else if(cm=='C') ke=transformC(d_a[idx].pkey);
      else if(cm=='D') ke=transformD(d_a[idx].pkey);
      d_a[idx].akey = ke;

} 
__global__ void transformerCon(struct work_entry *d_a)
{
      int idx = threadIdx.x;
      uint16_t  ke;
      if(d_a[idx].cmd=='A') ke = transformA(d_a[idx].akey);
      else if(d_a[idx].cmd=='B') ke = transformB(d_a[idx].akey);
      else if(d_a[idx].cmd=='C') ke = transformC(d_a[idx].akey);
      else if(d_a[idx].cmd=='D') ke = transformD(d_a[idx].akey);
      d_a[idx].pkey = ke;
}

int main()
{
  double producerTime = 0; //producer's time
  double consumerTime = 0; //consumer's time
  clock_t producerClock = 0; //producer's clock
  clock_t consumerClock = 0; //consumer's clock

  // assigning the threads based on the input file
  struct work_entry *data = (struct work_entry*) malloc(10000); 
  char lines[10];
  int num = 0;
  for(int i=0; fgets(lines, sizeof(lines), stdin); i++) 
  {
     char *token=strtok(lines, " \t");
     char cm = *token;
     if(*token=='A' || *token=='B' || *token=='C' || *token=='D')
     {
        token = strtok(NULL, " \t");
        if(atoi(token)>=0 && atoi(token)<=1000)
        {
          data[num].cmd = cm;
          data[num].pkey = atoi(token);
          data[num].akey = 0;
          num = num + 1;
        }
     }
  }
  
  struct work_entry *h_a; //host memory 
  struct work_entry *d_a; //device memory 

  //threads hierachy 
  int dimA = num; 
  int nblocks = 1;
  int tpb = dimA; 
  size_t memSize;

  memSize = nblocks*dimA*sizeof(struct work_entry);
  h_a = (struct work_entry*) malloc(memSize);
  hipMalloc((void**) &d_a, memSize);

  for(int i=0;i<num;i++)  //input data to the host memory 
  {
    h_a[i].cmd = data[i].cmd;
    h_a[i].pkey = data[i].pkey;
    h_a[i].akey = data[i].akey;
  }

  hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice); //copy input data to device

  dim3 dimGrid(nblocks);
  dim3 dimBlock(tpb);
  
  clock_t  pT;
  time_t  pStart, pEnd;
  time(&pStart);
  pT = clock();
                               
  transformerPro<<<dimGrid, dimBlock>>>(d_a); // producer  get worked

  time(&pEnd);
  pT = clock()-pT;
  producerTime = producerTime + difftime(pEnd, pStart);
  producerClock = producerClock + pT;


  //cudaMemcpy(h_a, d_a, memSize, cudaMemcpyDeviceToHost);

  //cudaMemcpy(d_a, h_a, memSize, cudaMemcpyHostToDevice);

  clock_t  cT;
  time_t  cStart, cEnd;
  time(&cStart);
  cT = clock();
                 
  transformerCon<<<dimGrid, dimBlock>>>(d_a);

  time(&cEnd);
  cT = clock()-cT;
  consumerTime = consumerTime + difftime(cEnd, cStart);
  consumerClock = consumerClock + cT;

  hipMemcpy(h_a, d_a, memSize, hipMemcpyDeviceToHost);

  for(int i=0;i<num;i++)
  {
    int idx = i % BUF_SIZE;
    printf("%d %c %d %d\n", idx, h_a[i].cmd, h_a[i].akey, h_a[i].pkey);
  }


 //output clock the producer thread and consumer thread take
 printf("total time of producer and consumer measured by clock(2):\n");
 printf("producer takes %d clicks %f seconds.\n", producerClock, (float)(producerClock)/CLOCKS_PER_SEC);
 printf("consumer takes %d clicks %f seconds.\n", consumerClock, (float)(consumerClock)/CLOCKS_PER_SEC);
 //output time the producer thread and consumer thread take
 printf("total time of producer and consumer measured by time(2):\n");
 printf("producer takes %f seconds.\n", producerTime);
 printf("consumer takes %f seconds.\n", consumerTime);

 }
