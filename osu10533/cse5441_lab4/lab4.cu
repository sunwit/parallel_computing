/*
   Name: Wei Sun (sun.1868@osu.edu)
   Class Section: CSE 5441 AU2018(27372) W F 12:45pm - 2:00pm
   Assignment Number: Programming Assignment 4
   Summary: Producer Consumer problem with GPU programming
   Submission: AU18
*/




#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<string.h>
#include<inttypes.h>
#include<time.h>

#define BUF_SIZE 5  //queue size is 5

struct work_entry //struct of queue
{
   char cmd;
   uint16_t pkey;
   uint16_t akey;
};


__device__ uint16_t transformA(uint16_t x); //transformA 
__device__ uint16_t transformB(uint16_t x); //transformB
__device__ uint16_t transformC(uint16_t x); //transformC
__device__ uint16_t transformD(uint16_t x); //transformD


__global__ void transformerPro(struct work_entry *d_a) 
{
     
      int idx = threadIdx.x;
      char cm = d_a[idx].cmd;
      if(cm=='A') d_a[idx].akey=transformA(d_a[idx].pkey);
      else if(cm=='B') d_a[idx].akey=transformB(d_a[idx].pkey);
      else if(cm=='C') d_a[idx].akey=transformC(d_a[idx].pkey);
      else if(cm=='D') d_a[idx].akey=transformD(d_a[idx].pkey);
}

__global__ void transformerCon(struct work_entry* d_a)
{
      int idx = threadIdx.x;
      uint16_t  ke;
      if(d_a[idx].cmd=='A') ke = transformA(d_a[idx].akey);
      else if(d_a[idx].cmd=='B') ke = transformB(d_a[idx].akey);
      else if(d_a[idx].cmd=='C') ke = transformC(d_a[idx].akey);
      else if(d_a[idx].cmd=='D') ke = transformD(d_a[idx].akey);
      d_a[idx].akey = ke;
}

int main()
{
  struct work_entry *data = (struct work_entry*) malloc(10000);
  char lines[10];
  int num = 0;
  for(int i=0; fgets(lines, sizeof(lines), stdin); i++) 
  {
     char *token=strtok(lines, " \t");
     char cm = *token;
     if(*token=='A' || *token=='B' || *token=='C' || *token=='D')
     {
        token = strtok(NULL, " \t");
        if(atoi(token)>=0 && atoi(token)<=1000)
        {
          data[num].cmd = cm;
          data[num].pkey = atoi(token);
          data[num].akey = 0;
          num = num + 1;
        }
     }
  }
  
  struct work_entry *h_a;
  struct work_entry *d_a;

  int dimA = num; 
  int nblocks = 1;
  int tpb = dimA; 
  size_t memSize;

  memSize = nblocks*dimA*sizeof(struct work_entry);
  h_a = (struct work_entry*) malloc(memSize);
  hipMalloc((void**) &d_a, memSize);

  for(int i=0;i<num;i++)
  {
    h_a[i].cmd = data[i].cmd;
    h_a[i].pkey = data[i].pkey;
    h_a[i].akey = data[i].akey;
  }

  hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice);

  dim3 dimGrid(nblocks);
  dim3 dimBlock(tpb);
  
  transformerPro<<<dimGrid, dimBlock>>>(d_a); 

  hipMemcpy(h_a, d_a, memSize, hipMemcpyDeviceToHost);


  transformerCon<<<dimGrid, dimBlock>>>(d_a); 


  for(int i=0; i<dimA; i++)
  {
   int idx = i%BUF_SIZE;
   printf("%d %c %d %d\n", idx, h_a[idx].cmd, h_a[idx].pkey, h_a[idx].akey);
  }
 
 }
