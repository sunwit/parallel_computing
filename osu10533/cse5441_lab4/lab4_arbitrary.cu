/*
   Name: Wei Sun (sun.1868@osu.edu)
   Class Section: CSE 5441 AU2018(27372) W F 12:45pm - 2:00pm
   Assignment Number: Programming Assignment 4
   Summary: Producer Consumer problem with GPU programming
   Submission: AU18

   Main Idea: this is the arbitrary version of producer-consumer with GPU programming. 
   the most time consuming part of producer and consumer is transforms. 
   then, we let GPU to do this most time consuming part. but, as you can see 
   we do these transforms in an arbitary order. so, version 1 is less efficient
   than version 2 which can reorder the transform part for GPU. 
*/


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<string.h>
#include<inttypes.h>
#include<time.h>

#define BUF_SIZE 5  //queue size is 5

struct work_entry //struct of queue
{
   char cmd;  // save letters
   uint16_t pkey; //save the key before producer transforms it
   uint16_t akey; //save the key after consumer transforms it

};


__device__ uint16_t transformA(uint16_t x); //transformA 
__device__ uint16_t transformB(uint16_t x); //transformB
__device__ uint16_t transformC(uint16_t x); //transformC
__device__ uint16_t transformD(uint16_t x); //transformD


__global__ void transformerPro(struct work_entry *d_a, int Asize) //producer gets started  in gpu kernel
{
      for(int step=0; step<Asize; step+=blockDim.x*gridDim.x) //the thread will transform whatever they get in an arbitrary order
      {
         int idx = step+(blockIdx.x*blockDim.x)+threadIdx.x;
         if(d_a[idx].cmd=='A') d_a[idx].akey = transformA(d_a[idx].pkey);
         else if(d_a[idx].cmd=='B') d_a[idx].akey = transformB(d_a[idx].pkey);
         else if(d_a[idx].cmd=='C') d_a[idx].akey = transformC(d_a[idx].pkey);
         else if(d_a[idx].cmd=='D') d_a[idx].akey = transformD(d_a[idx].pkey);
      }

} 

__global__ void transformerCon(struct work_entry *d_a, int Asize) //consumer gets started in gpu kernel
{
      for(int step=0; step<Asize;step+=blockDim.x*gridDim.x) //the thread will transform whatever they get in an arbitrary order
      {
        int idx = step+(blockIdx.x*blockDim.x)+threadIdx.x;
        uint16_t  ke;
        if(d_a[idx].cmd=='A') ke = transformA(d_a[idx].akey);
        else if(d_a[idx].cmd=='B') ke = transformB(d_a[idx].akey);
        else if(d_a[idx].cmd=='C') ke = transformC(d_a[idx].akey);
        else if(d_a[idx].cmd=='D') ke = transformD(d_a[idx].akey);
        d_a[idx].pkey = ke;
      }
}

int main()
{
  double producerTime = 0; //producer's time
  double consumerTime = 0; //consumer's time
  clock_t producerClock = 0; //producer's clock
  clock_t consumerClock = 0; //consumer's clock

  
  struct work_entry *h_a; //host memory 
  struct work_entry *d_a; //device memory 


  struct work_entry *data = (struct work_entry*) malloc(10000*sizeof(struct work_entry));  // read data from the file
  char lines[10];
  int num = 0;
  for(int i=0; fgets(lines, sizeof(lines), stdin); i++) 
  {
     char *token=strtok(lines, " \t");
     char cm = *token;
     if(*token=='A' || *token=='B' || *token=='C' || *token=='D')
     {
        token = strtok(NULL, " \t");
        if(atoi(token)>=0 && atoi(token)<=1000)
        {
          data[num].cmd = cm;
          data[num].pkey = atoi(token);
          num = num + 1;
        }
     }
  }
 

  //threads hierachy 
  int nblocks = 1;
  int tpb = 1024; 
  size_t memSize;

  memSize = num*sizeof(struct work_entry);
  h_a = (struct work_entry*) malloc(memSize); //allocate memory for host

  hipMalloc((void**) &d_a, memSize); //alocate memory for device


  for(int i=0;i<num;i++)  //input data to the host memory 
  {
    
    h_a[i].cmd = data[i].cmd;
    h_a[i].pkey = data[i].pkey;
  }
  hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice); //copy input data to device

  dim3 dimGrid(nblocks);
  dim3 dimBlock(tpb);

  // time and clock calculation for producer
  clock_t  pT;
  time_t  pStart, pEnd;
  time(&pStart);
  pT = clock();
  //launch the gpu kerenel to process data for producer                            
  transformerPro<<<dimGrid, dimBlock>>>(d_a, num); // the most time conumming part in producer side
  hipMemcpy(h_a, d_a, memSize, hipMemcpyDeviceToHost); //copy back the data from gpu kernel to host memory


  time(&pEnd);
  pT = clock()-pT;
  producerTime = producerTime + difftime(pEnd, pStart);
  producerClock = producerClock + pT;

  // time and clock calculation for consumer
  clock_t  cT;
  time_t  cStart, cEnd;
  time(&cStart);
  cT = clock();
  //lauch the gpu kernel to process data for consumer
  transformerCon<<<dimGrid, dimBlock>>>(d_a, num); //the most time consumming part in consumer side 
               
  hipMemcpy(h_a, d_a, memSize, hipMemcpyDeviceToHost); //copy back the data from the device to host 
  for(int i=0;i<num;i++) //consumer prints the result 
  {
    int idx = i % BUF_SIZE;
    printf("%d %c %d %d\n", idx, h_a[i].cmd, h_a[i].pkey, h_a[i].akey);
  }
  time(&cEnd);
  cT = clock()-cT;
  consumerTime = consumerTime + difftime(cEnd, cStart);
  consumerClock = consumerClock + cT;
 //output clock the producer thread and consumer thread take
 printf("total time of producer and consumer measured by clock(2):\n");
 printf("producer takes %d clicks %f seconds.\n", producerClock, (float)(producerClock)/CLOCKS_PER_SEC);
 printf("consumer takes %d clicks %f seconds.\n", consumerClock, (float)(consumerClock)/CLOCKS_PER_SEC);
 //output time the producer thread and consumer thread take
 printf("total time of producer and consumer measured by time(2):\n");
 printf("producer takes %f seconds.\n", producerTime);
 printf("consumer takes %f seconds.\n", consumerTime);
 }
