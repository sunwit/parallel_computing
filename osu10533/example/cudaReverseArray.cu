
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void reverseArrayBlock(int *inArray, int *outArray) 
{
   outArray[blockDim.x-1-threadIdx.x] = inArray[threadIdx.x];
}

int main()
{
  int *h_a;
  int *d_a;
  int *d_b;
  int dimA=256;
  int nblocks = 1;
  int tpb = dimA;
  size_t memSize;
  memSize = nblocks*tpb*sizeof(int);
  h_a = (int*) malloc(memSize);
  hipMalloc((void**)&d_a, memSize);
  hipMalloc((void**)&d_b, memSize);
  
  for(int i=0;i<dimA;i++)
  {
     h_a[i] = i;
  }
  hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice);
  dim3 dimGrid(nblocks);
  dim3 dimBlock(tpb);
  reverseArrayBlock<<<dimGrid, dimBlock>>>(d_a, d_b);  

  hipMemcpy(h_a, d_b, memSize, hipMemcpyDeviceToHost);
  
  bool good_results = true;
  for(int i=0;(i<dimA)&& good_results;i++)
  {
     if(h_a[dimA-1-i]!=i)
     {
       printf("%s\n", "oops");
       good_results=false;
     }
     else
     {
       printf("yes \n");
     }
  }

}
