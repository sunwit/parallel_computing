/*
   Name: Wei Sun (sun.1868@osu.edu)
   Class Section: CSE 5441 AU2018(27372) W F 12:45pm - 2:00pm
   Assignment Number: Programming Assignment 4
   Summary: Producer Consumer problem with GPU programming
   Submission: AU18
   
   Main Idea: this is version 2 of producer consumer problem with gpu programming.
   comparing with the version 1, in the version 2, we let the gpu kernel to perform 
   the transforms without if-else statements.  In the gpu kernel, using while statement
   we do transformA, transformB, transformC and transformD sequentially.  So, we can 
   improve the performance in comparision to the version 1 because of thread 
   divergence.  
*/


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<string.h>
#include<inttypes.h>
#include<time.h>

#define BUF_SIZE 5  //queue size is 5

struct work_entry //struct of queue
{
   char cmd;  // save letters
   uint16_t pkey; //save the key before producer transforms it
   uint16_t akey; //save the key after consumer transforms it

};


__device__ uint16_t transformA(uint16_t x); //transformA 
__device__ uint16_t transformB(uint16_t x); //transformB
__device__ uint16_t transformC(uint16_t x); //transformC
__device__ uint16_t transformD(uint16_t x); //transformD


__global__ void transformerPro(struct work_entry *d_a, int Asize) //producer gets started  in gpu kernel
{
      int step = 0;
      int idx = step+(blockIdx.x*blockDim.x)+threadIdx.x;
      while(step<Asize && d_a[idx].cmd=='A')  //doing transformA
      {
        d_a[idx].akey = transformA(d_a[idx].pkey);
        step = step + (blockDim.x*gridDim.x);
        idx = step + (blockIdx.x*blockDim.x) + threadIdx.x;
      }

      while(step<Asize && d_a[idx].cmd=='B') //doing transformB
      {
        d_a[idx].akey = transformB(d_a[idx].pkey);
        step = step + (blockDim.x*gridDim.x);
        idx = step + (blockIdx.x*blockDim.x) + threadIdx.x;
      }

      while(step<Asize && d_a[idx].cmd=='C') //doing transformC
      {
        d_a[idx].akey = transformC(d_a[idx].pkey);
        step = step + (blockDim.x*gridDim.x);
        idx = step + (blockIdx.x*blockDim.x) + threadIdx.x;
      }

      while(step<Asize && d_a[idx].cmd=='D') //doing transformD
      {
        d_a[idx].akey = transformD(d_a[idx].pkey);
        step = step + (blockDim.x*gridDim.x);
        idx = step + (blockIdx.x*blockDim.x) + threadIdx.x;
      }

} 





__global__ void transformerCon(struct work_entry *d_a, int Asize) //consumer gets started in gpu kernel
{
      int step = 0;
      int idx = step+(blockIdx.x*blockDim.x)+threadIdx.x;
      while(step<Asize && d_a[idx].cmd=='A') //doing transformA
      {
        d_a[idx].pkey = transformA(d_a[idx].akey);
        step = step + (blockDim.x*gridDim.x);
        idx = step + (blockIdx.x*blockDim.x) + threadIdx.x;
      }

      while(step<Asize && d_a[idx].cmd=='B') //doing transformB
      {
        d_a[idx].pkey = transformB(d_a[idx].akey);
        step = step + (blockDim.x*gridDim.x);
        idx = step + (blockIdx.x*blockDim.x) + threadIdx.x;
      }

      while(step<Asize && d_a[idx].cmd=='C') //doing transformC
      {
        d_a[idx].pkey = transformC(d_a[idx].akey);
        step = step + (blockDim.x*gridDim.x);
        idx = step + (blockIdx.x*blockDim.x) + threadIdx.x;
      }

      while(step<Asize && d_a[idx].cmd=='D') //doing transformD
      {
        d_a[idx].pkey = transformD(d_a[idx].akey);
        step = step + (blockDim.x*gridDim.x);
        idx = step + (blockIdx.x*blockDim.x) + threadIdx.x;
      }
}


int main()
{
  double producerTime = 0; //producer's time
  double consumerTime = 0; //consumer's time
  clock_t producerClock = 0; //producer's clock
  clock_t consumerClock = 0; //consumer's clock

  
  struct work_entry *h_a; //host memory 
  struct work_entry *d_a; //device memory 


  struct work_entry *data = (struct work_entry*) malloc(10000*sizeof(struct work_entry));  // read data from the file
  char lines[10];
  int num = 0;
  int num_a = 0;
  int num_b = 0;
  int num_c = 0;
  int num_d = 0;
  for(int i=0; fgets(lines, sizeof(lines), stdin); i++) 
  {
     char *token=strtok(lines, " \t");
     char cm = *token;
     if(*token=='A' || *token=='B' || *token=='C' || *token=='D')
     {
        token = strtok(NULL, " \t");
        if(atoi(token)>=0 && atoi(token)<=1000)
        {
          data[num].cmd = cm;
          if(cm=='A') num_a = num_a + 1;
          if(cm=='B') num_b = num_b + 1;
          if(cm=='C') num_c = num_c + 1;
          if(cm=='D') num_d = num_d + 1;
          data[num].pkey = atoi(token);
          num = num + 1;
        }
     }
  }
 

  //threads hierachy 
  int nblocks = 1;
  int tpb = 1024; 
  size_t memSize;

  memSize = num*sizeof(struct work_entry);
  h_a = (struct work_entry*) malloc(memSize);
  hipMalloc((void**) &d_a, memSize);


  int ia=0, ib=num_a, ic=ib+num_b, id=ic+num_c;
  for(int i=0;i<num;i++) //pre-processing data into four groups such that gpu kernel can process them conveniently.
  {
     if(data[i].cmd=='A')  //the order of data is going from A, B, C to D.
     {
       h_a[ia].cmd  = 'A';
       h_a[ia].pkey = data[i].pkey; 
       ia = ia + 1;
     }
     else if(data[i].cmd=='B')
     {
       h_a[ib].cmd = 'B';
       h_a[ib].pkey = data[i].pkey; 
       ib = ib + 1;
     }
     else if(data[i].cmd=='C')
     {
       h_a[ic].cmd = 'C';
       h_a[ic].pkey = data[i].pkey; 
       ic = ic + 1;
     }
     else if(data[i].cmd=='D')
     {
       h_a[id].cmd = 'D';
       h_a[id].pkey = data[i].pkey; 
       id = id + 1;
     }
  }

  
  hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice); //copy input data to device

  dim3 dimGrid(nblocks);
  dim3 dimBlock(tpb);

  // time and clock calculation for producer
  clock_t  pT;
  time_t  pStart, pEnd;
  time(&pStart);
  pT = clock();
  //launch the gpu kernel for producer to transforming                             
  transformerPro<<<dimGrid, dimBlock>>>(d_a, num); // the most time conumming part in producer side
  hipMemcpy(h_a, d_a, memSize, hipMemcpyDeviceToHost);



  time(&pEnd);
  pT = clock()-pT;
  producerTime = producerTime + difftime(pEnd, pStart);
  producerClock = producerClock + pT;

  // time and clock calculation for consumer
  clock_t  cT;
  time_t  cStart, cEnd;
  time(&cStart);
  cT = clock();
  //launch the gpu kernel for consumer to do transforming
  transformerCon<<<dimGrid, dimBlock>>>(d_a, num); //the most time consumming part in consumer side 
  hipMemcpy(h_a, d_a, memSize, hipMemcpyDeviceToHost); //copy data from gpu kernel to host



  for(int i=0;i<num;i++) //consumer prints the result 
  {
    int idx = i % BUF_SIZE;
    printf("%d %c %d %d\n", idx, h_a[i].cmd, h_a[i].pkey, h_a[i].akey);
  }

  time(&cEnd);
  cT = clock()-cT;
  consumerTime = consumerTime + difftime(cEnd, cStart);
  consumerClock = consumerClock + cT;


 //output clock the producer thread and consumer thread take
 printf("total time of producer and consumer measured by clock(2):\n");
 printf("producer takes %d clicks %f seconds.\n", producerClock, (float)(producerClock)/CLOCKS_PER_SEC);
 printf("consumer takes %d clicks %f seconds.\n", consumerClock, (float)(consumerClock)/CLOCKS_PER_SEC);
 //output time the producer thread and consumer thread take
 printf("total time of producer and consumer measured by time(2):\n");
 printf("producer takes %f seconds.\n", producerTime);
 printf("consumer takes %f seconds.\n", consumerTime);
 }
